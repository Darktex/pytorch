
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Sigmoid.cu"
#else

#include "../common.h"

void THNN_(Sigmoid_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(sigmoid)(state, output, input);
}

void THNN_(Sigmoid_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, SigmoidGradInputOp<real>());
}

#endif
